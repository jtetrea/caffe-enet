#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/spatial_dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SpatialDropoutForward(const int n, const int h, const int w, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    int current_channel = index % (h * w);
    out[index] = in[index] * (mask[current_channel] > threshold) * scale;
  }
}

template <typename Dtype>
void SpatialDropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int channels = bottom[0]->shape(1); // number of channels
  const int height = bottom[0]->shape(2);
  const int width = bottom[0]->shape(3);
  if (this->phase_ == TRAIN || this->layer_param_.spatial_dropout_param().sample_weights_test()) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());

    caffe_gpu_rng_uniform(channels, mask);
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    SpatialDropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, height, width, bottom_data, mask, uint_thres_, scale_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void SpatialDropoutBackward(const int n, const int h, const int w, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    int current_channel = index % (h * w);
    out_diff[index] = in_diff[index] * (mask[current_channel] > threshold) * scale;
  }
}

template <typename Dtype>
void SpatialDropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN || this->layer_param_.spatial_dropout_param().sample_weights_test()) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
      const int channels = bottom[0]->shape(1); // number of channels
      const int height = bottom[0]->shape(2);
      const int width = bottom[0]->shape(3);
      // NOLINT_NEXT_LINE(whitespace/operators)
      SpatialDropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, height, width, top_diff, mask, uint_thres_, scale_, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SpatialDropoutLayer);

}  // namespace caffe
